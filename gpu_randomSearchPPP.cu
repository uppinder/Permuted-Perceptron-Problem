#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include "HandleError.h"

#define M 801 
#define N 817
#define K 15 // Choose K <= 16 < N.
#define G 30

void generateData(int A[M][N],int Y[N],int S[M],int H[N+1]);
void matrixMul(int A[M][N],int Y[N],int S[M]);

__global__ void computeMinCost(int *dev_A,int *dev_H);


int main(int argc,char* argv[]) {

	int A[M][N],Y[N],S[M],H[N+1];
	int *dev_A,*dev_H;

	int size_A = M * N * sizeof(int);
	int size_H = (N+1) * sizeof(int);

	
	/*
		Randomly generate A,Y and S. Compute H.
	 */
	generateData(A,Y,S,H);


	hipMalloc((void**)&dev_A,size_A);
	hipMalloc((void**)&dev_H,size_H);

	hipMemcpy(dev_A,A,size_A,hipMemcpyHostToDevice);
	hipMemcpy(dev_H,H,size_H,hipMemcpyHostToDevice);

	/*
		Launch kernel ... for calculating A*X for a
		particular block. In total 2^K blocks required.
	 */
	// Number of blocks required.

	float timeTaken;
	hipEvent_t start,stop;

	HANDLE_ERROR( hipEventCreate(&start));
	HANDLE_ERROR( hipEventCreate(&stop));
	HANDLE_ERROR( hipEventRecord(start, 0));

	const int numOfBlocks = 1 << K;
	computeMinCost<<< numOfBlocks, N >>>(dev_A,dev_H);
	
	HANDLE_ERROR( hipEventRecord(stop, 0));
	HANDLE_ERROR( hipEventSynchronize(stop));
	HANDLE_ERROR( hipEventElapsedTime(&timeTaken, start, stop));
	
	printf("\nTime taken: %0.10fs\n\n",timeTaken/1000);	

	hipFree(dev_A);
	hipFree(dev_H);

	return 0;
}


__global__ void computeMinCost(int *A,int *H) {

	/*
		Get block number and calulate X. Compute S_x = AX. After
		that, compute S_xx for a particular thread in O(N). 
		( By flipping one bit(acc to thread no.)). Compute cost.  
	 */
	
	// Calculate X for a particular block.
		__shared__ int X[N],blockNumber;

		int idx = threadIdx.x;
		blockNumber = blockIdx.x;
		
		if(idx < N) {

			X[idx] = -1;
			if(idx < K)
				X[idx] = ( blockNumber & (1<<idx)) ? 1:-1; 	
		
		}

	__syncthreads();
	
	//Calculate S = AX for a particular block.
		__shared__ int S[M];

		int k,sum = 0;

		if(idx < M) {
			
			for(k = 0;k < N;k++)
				sum += A[idx * N + k] * X[k];

			S[idx] = sum; 
		}

	__syncthreads();

	// Now, on a thread basis, calculate cost.
	if(idx >= K && idx < N) {

		//Compute S_idx for a thread.
			int i,S_idx[M];
			for(i = 0;i < M;i++)
				S_idx[i] = S[i] + 2 * A[i * N + idx] * X[idx];

		//Calculate cost.
			int H_idx[N+1];
			
			for(i = 0; i <= N; i++)
				H_idx[i] = 0;

			for(i = 0;i < M;i++) {
				if(S_idx[i] >= 0)
					H_idx[S_idx[i]]++;
			}

			//1-Hamming Distance
			int hamDist = 0;
			for(i = 0;i < M;i++)
				hamDist += abs(S_idx[i]) - S_idx[i];

			//Histogram variation
			int histVar = 0;
			for(i = 0;i <= N;i++)
				histVar += abs(H[i] - H_idx[i]);

			int cost = G*hamDist + histVar;

			//printf("%d\n",cost);

	}

}


/*
	Generate random A and Y. Compute AY.
	If ith entry of AY is -ve, flip all 
	entries of ith row of A. When all 
	entries of AY are +ve, calculated A,Y,S
	are a data set.
 */
void generateData(int A[M][N],int Y[N],int S[M],int H[N+1]) {

	srand(time(NULL));
	int i,j;

	for(i = 0;i < M;i++)
		for(j = 0;j < N;j++) {

			A[i][j] = (rand()%2 == 0) ? -1:1; 
		}

	for(i = 0;i < N;i++) {
		Y[i] = (rand()%2 == 0) ? -1:1;
	}

	matrixMul(A,Y,S);

	for(i = 0;i < M;i++) {
		if(S[i] < 0) { 
			for(j = 0;j < N;j++) {
				A[i][j] = -A[i][j];
			}
			S[i] = -S[i];
		}
	}

	for(i = 0;i <= N;i++)
		H[i] = 0;

	for(i = 0;i < M;i++)
		H[S[i]]++;

}

void matrixMul(int A[M][N],int Y[N],int S[M]) {

	int i,k;

	for(i = 0;i < M;i++)  {
		S[i] = 0;
		for(k = 0;k < N ;k++)
			S[i] += A[i][k] * Y[k];
	}
		
}

